#include "hip/hip_runtime.h"
#include "ex1.h"

#define COLOR_VALUES 256
#define THREADS_PER_BLOCK 1024

// Requires atleast <size of arr> threads
__device__
void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;

    for(int stride = 1; stride < arr_size; stride *= 2){
        if(tid >= stride & tid < arr_size){
            increment = arr[tid - stride];
        }
        __syncthreads();
        if(tid >= stride & tid < arr_size){
            arr[tid] += increment;
        }
        __syncthreads();
    }
    return;
}

/**
 * Calculates the histogram of a single tile of an image
 * @param histogram int array of size [COLOR_VALUES].
 * @param image_in 2D char array of the input image.
 * @param tile_row_num the index of the current tile's row in the image.
 * @param tile_col_num the index of the current tile's column in the image.
 */
__device__
void calc_tile_histogram(int* histogram, uchar* image_in, int tile_row_num, int tile_col_num)
{
    int pixel_value = 0;
    int index_in_img = 0;

    const int tid = threadIdx.x;
    const int rows_group_size = blockDim.x / TILE_WIDTH;
    const int row_index = tile_row_num * TILE_WIDTH + tid / TILE_WIDTH;
    const int col_index = tile_col_num * TILE_WIDTH + tid % TILE_WIDTH;
    
    for(int i = 0 ; i < TILE_WIDTH ; i+=rows_group_size)
    {
        index_in_img = (row_index + i) * IMG_WIDTH + col_index;
        pixel_value = image_in[index_in_img];
        if (row_index + i < (tile_row_num + 1) * TILE_WIDTH)
            atomicAdd(&histogram[pixel_value], 1);
    }  
}

/**
 * Calculates the map for the current tile of the histogram equalization
 * @param maps 3D array of size [TILE_COUNT][TILE_COUNT][COLOR_VALUES] that maps each tiles 
 *             gray values from before the equalization to after it.
 * @param tile_row_num the index of the current tile's row in the image.
 * @param tile_col_num the index of the current tile's column in the image.
 * @param CDF_func int array of the CDF function of the tile's histogram.
 */
__device__
void calc_tile_map(uchar* maps, int tile_row_num, int tile_col_num, int* CDF_func)
{
    const int tid = threadIdx.x;
    int start_index_in_map = tile_row_num * TILE_COUNT * COLOR_VALUES + tile_col_num * COLOR_VALUES;
    
    const int numThreads = blockDim.x;
    int work_per_thread = COLOR_VALUES / numThreads;
    if (COLOR_VALUES % numThreads != 0)
        work_per_thread++;
    
    for(int i = 0; i < work_per_thread; i++)
        if(tid + i * numThreads < COLOR_VALUES){
            maps[start_index_in_map + tid + i * numThreads] = float(CDF_func[tid + i * numThreads]) * (COLOR_VALUES - 1) / (TILE_WIDTH * TILE_WIDTH);
        }
}

/**
 * Initiate array with an input value
 * @param arr int array
 * @param length the length of the array
 * @param value the value to initiate the array with
 */
__device__
void array_initiate(int* arr, int length, int value)
{
    const int tid = threadIdx.x;
    
    const int numThreads = blockDim.x;
    int work_per_thread = COLOR_VALUES / numThreads;
    if (COLOR_VALUES % numThreads != 0)
        work_per_thread++;
    
    for(int i = 0; i < work_per_thread; i++)
        if(tid + i * numThreads < length)
            arr[tid + i * numThreads] = value;
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__ 
void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

__global__ void process_image_kernel(uchar *all_in, uchar *all_out, uchar *maps) {
    
    __shared__ int histogram[sizeof(int) * COLOR_VALUES];
    const int offset_in_img = IMG_HEIGHT * IMG_WIDTH * blockIdx.x;
    const int offset_in_maps = COLOR_VALUES * TILE_COUNT * TILE_COUNT * blockIdx.x;

    for(int tile_row = 0; tile_row < TILE_COUNT; tile_row++)
    {
        for(int tile_col = 0; tile_col < TILE_COUNT; tile_col++){
            array_initiate(histogram, COLOR_VALUES, 0);
            __syncthreads();

            calc_tile_histogram(histogram, all_in + offset_in_img, tile_row, tile_col);
            __syncthreads();

            prefix_sum(histogram, COLOR_VALUES);
            __syncthreads();

            calc_tile_map(maps + offset_in_maps, tile_row, tile_col, histogram);
            __syncthreads();
        }
    }

    interpolate_device(maps + offset_in_maps, all_in + offset_in_img, all_out + offset_in_img);
    __syncthreads();

    return; 
}

/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context {
    uchar* image_in;   // size of [IMG_HEIGHT][IMG_WIDTH]
    uchar* image_out;  // size of [IMG_HEIGHT][IMG_WIDTH]
    uchar* tiles_maps; // size of [TILE_COUNT][TILE_COUNT][COLOR_VALUES]
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context *task_serial_init()
{
    auto context = new task_serial_context;

    //Allocate GPU memory for a single input image, a single output image, and maps
    CUDA_CHECK(hipMalloc((void**)&context->image_in, sizeof(char) * IMG_HEIGHT * IMG_WIDTH));
    CUDA_CHECK(hipMalloc((void**)&context->image_out, sizeof(char) * IMG_HEIGHT * IMG_WIDTH));
    CUDA_CHECK(hipMalloc((void**)&context->tiles_maps, sizeof(char) * TILE_COUNT * TILE_COUNT * COLOR_VALUES));

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_in, uchar *images_out)
{
    uchar* current_image_in;
    uchar* current_image_out;

    for(int i = 0; i < N_IMAGES; i++){
        current_image_in = images_in + IMG_HEIGHT * IMG_WIDTH * i;
        current_image_out = images_out + IMG_HEIGHT * IMG_WIDTH * i;

        CUDA_CHECK(hipMemcpy(context->image_in, current_image_in, sizeof(uchar) * IMG_HEIGHT * IMG_WIDTH, hipMemcpyHostToDevice));
        process_image_kernel<<<1, THREADS_PER_BLOCK>>>(context->image_in, context->image_out, context->tiles_maps);
        CUDA_CHECK(hipMemcpy(current_image_out, context->image_out, sizeof(uchar) * IMG_HEIGHT * IMG_WIDTH, hipMemcpyDeviceToHost));
    }

}

/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    CUDA_CHECK(hipFree(context->image_in));
    CUDA_CHECK(hipFree(context->image_out));
    CUDA_CHECK(hipFree(context->tiles_maps));

    delete(context);
}

/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context {
    uchar* images_in;  // size of [N_IMAGES][IMG_HEIGHT][IMG_WIDTH]
    uchar* images_out; // size of [N_IMAGES][IMG_HEIGHT][IMG_WIDTH]
    uchar* tiles_maps; // size of [N_IMAGES][TILE_COUNT][TILE_COUNT][COLOR_VALUES]
};

/* Allocate GPU memory for all the input images, output images, and maps.
 * 
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;

    //Allocate GPU memory for a all input images, all output images, and all maps
    CUDA_CHECK(hipMalloc((void**)&context->images_in, sizeof(char) * N_IMAGES * IMG_HEIGHT * IMG_WIDTH));
    CUDA_CHECK(hipMalloc((void**)&context->images_out, sizeof(char) * N_IMAGES * IMG_HEIGHT * IMG_WIDTH));
    CUDA_CHECK(hipMalloc((void**)&context->tiles_maps, sizeof(char) * N_IMAGES * TILE_COUNT * TILE_COUNT * COLOR_VALUES));

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_in, uchar *images_out)
{
    CUDA_CHECK(hipMemcpy(context->images_in, images_in, sizeof(uchar) * N_IMAGES * IMG_HEIGHT * IMG_WIDTH, hipMemcpyHostToDevice));
    process_image_kernel<<<N_IMAGES, THREADS_PER_BLOCK>>>(context->images_in, context->images_out, context->tiles_maps);
    CUDA_CHECK(hipMemcpy(images_out, context->images_out, sizeof(uchar) * N_IMAGES * IMG_HEIGHT * IMG_WIDTH, hipMemcpyDeviceToHost));
}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    CUDA_CHECK(hipFree(context->images_in));
    CUDA_CHECK(hipFree(context->images_out));
    CUDA_CHECK(hipFree(context->tiles_maps));

    delete(context);
}
