#include "hip/hip_runtime.h"
#include "ex2.h"
#include <cuda/atomic>

#define COLOR_VALUES 256
#define THREADS_PER_BLOCK 1024
#define EMPTY_STREAM -1

// Requires atleast <size of arr> threads
__device__ void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;

    for(int stride = 1; stride < arr_size; stride *= 2){
        if(tid >= stride & tid < arr_size){
            increment = arr[tid - stride];
        }
        __syncthreads();
        if(tid >= stride & tid < arr_size){
            arr[tid] += increment;
        }
        __syncthreads();
    }
    return;
}

/**
 * Calculates the histogram of a single tile of an image
 * @param histogram int array of size [COLOR_VALUES].
 * @param image_in 2D char array of the input image.
 * @param tile_row_num the index of the current tile's row in the image.
 * @param tile_col_num the index of the current tile's column in the image.
 */
__device__
void calc_tile_histogram(int* histogram, uchar* image_in, int tile_row_num, int tile_col_num)
{
    int pixel_value = 0;
    int index_in_img = 0;

    const int tid = threadIdx.x;
    const int rows_group_size = blockDim.x / TILE_WIDTH;
    const int row_index = tile_row_num * TILE_WIDTH + tid / TILE_WIDTH;
    const int col_index = tile_col_num * TILE_WIDTH + tid % TILE_WIDTH;
    
    for(int i = 0 ; i < TILE_WIDTH ; i+=rows_group_size)
    {
        index_in_img = (row_index + i) * IMG_WIDTH + col_index;
        pixel_value = image_in[index_in_img];
        if (row_index + i < (tile_row_num + 1) * TILE_WIDTH)
            atomicAdd(&histogram[pixel_value], 1);
    }  
}

/**
 * Calculates the map for the current tile of the histogram equalization
 * @param maps 3D array of size [TILE_COUNT][TILE_COUNT][COLOR_VALUES] that maps each tiles 
 *             gray values from before the equalization to after it.
 * @param tile_row_num the index of the current tile's row in the image.
 * @param tile_col_num the index of the current tile's column in the image.
 * @param CDF_func int array of the CDF function of the tile's histogram.
 */
__device__
void calc_tile_map(uchar* maps, int tile_row_num, int tile_col_num, int* CDF_func)
{
    const int tid = threadIdx.x;
    int start_index_in_map = tile_row_num * TILE_COUNT * COLOR_VALUES + tile_col_num * COLOR_VALUES;
    
    const int numThreads = blockDim.x;
    int work_per_thread = COLOR_VALUES / numThreads;
    if (COLOR_VALUES % numThreads != 0)
        work_per_thread++;
    
    for(int i = 0; i < work_per_thread; i++)
        if(tid + i * numThreads < COLOR_VALUES){
            maps[start_index_in_map + tid + i * numThreads] = float(CDF_func[tid + i * numThreads]) * (COLOR_VALUES - 1) / (TILE_WIDTH * TILE_WIDTH);
        }
}

/**
 * Initiate array with an input value
 * @param arr int array
 * @param length the length of the array
 * @param value the value to initiate the array with
 */
__device__
void array_initiate(int* arr, int length, int value)
{
    const int tid = threadIdx.x;
    
    const int numThreads = blockDim.x;
    int work_per_thread = COLOR_VALUES / numThreads;
    if (COLOR_VALUES % numThreads != 0)
        work_per_thread++;
    
    for(int i = 0; i < work_per_thread; i++)
        if(tid + i * numThreads < length)
            arr[tid + i * numThreads] = value;
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__
 void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

__device__
void process_image(uchar *in, uchar *out, uchar* maps) {
    // TODO complete according to hw1
    __shared__ int histogram[sizeof(int) * COLOR_VALUES];

    for(int tile_row = 0; tile_row < TILE_COUNT; tile_row++)
    {
        for(int tile_col = 0; tile_col < TILE_COUNT; tile_col++){
            array_initiate(histogram, COLOR_VALUES, 0);
            __syncthreads();

            calc_tile_histogram(histogram, in, tile_row, tile_col);
            __syncthreads();

            prefix_sum(histogram, COLOR_VALUES);
            __syncthreads();

            calc_tile_map(maps, tile_row, tile_col, histogram);
            __syncthreads();
        }
    }

    interpolate_device(maps, in, out);
    __syncthreads();

    return; 
}

__global__
void process_image_kernel(uchar *in, uchar *out, uchar* maps){
    process_image(in, out, maps);
}

class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
    hipStream_t server_streams[STREAM_COUNT];
    int streams_image_id[STREAM_COUNT]; // size of [STREAM_COUNT]. Indicates what image_id is running on each stream.
    uchar* images_in[STREAM_COUNT];     // size of [STREAM_COUNT][IMG_HEIGHT][IMG_WIDTH]
    uchar* images_out[STREAM_COUNT];    // size of [STREAM_COUNT][IMG_HEIGHT][IMG_WIDTH]
    uchar* tiles_maps[STREAM_COUNT];    // size of [STREAM_COUNT][TILE_COUNT][TILE_COUNT][COLOR_VALUES]

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
        for (int i = 0; i < STREAM_COUNT; i++) {
            hipStreamCreate(&this->server_streams[i]);
            this->streams_image_id[i] = EMPTY_STREAM;
            CUDA_CHECK(hipMalloc((void**)&this->images_in[i], sizeof(char) * IMG_HEIGHT * IMG_WIDTH));
            CUDA_CHECK(hipMalloc((void**)&this->images_out[i], sizeof(char) * IMG_HEIGHT * IMG_WIDTH));
            CUDA_CHECK(hipMalloc((void**)&this->tiles_maps[i], sizeof(char) * TILE_COUNT * TILE_COUNT * COLOR_VALUES));
        }
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
        for (int i = 0; i < STREAM_COUNT; i++) {
            hipStreamDestroy(this->server_streams[i]);
            CUDA_CHECK(hipFree(this->images_in[i]));
            CUDA_CHECK(hipFree(this->images_out[i]));
            CUDA_CHECK(hipFree(this->tiles_maps[i]));
        }
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
        for(int i = 0; i < STREAM_COUNT; i++){
            if(this->streams_image_id[i] == EMPTY_STREAM){
                this->streams_image_id[i] = img_id;
                CUDA_CHECK(hipMemcpyAsync(this->images_in[i], img_in, sizeof(uchar) * IMG_HEIGHT * IMG_WIDTH, hipMemcpyHostToDevice, this->server_streams[i]));
                process_image_kernel<<<1, THREADS_PER_BLOCK, 0, this->server_streams[i]>>>(this->images_in[i], this->images_out[i], this->tiles_maps[i]);
                CUDA_CHECK(hipMemcpyAsync(img_out, this->images_out[i], sizeof(uchar) * IMG_HEIGHT * IMG_WIDTH, hipMemcpyDeviceToHost, this->server_streams[i]));
                return true;
            }
        }

        return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) streams for any completed requests.
        for(int i = 0; i < STREAM_COUNT; i++){
            if(this->streams_image_id[i] != EMPTY_STREAM){
                hipError_t status = hipStreamQuery(this->server_streams[i]); // TODO query diffrent stream each iteration
                switch (status) {
                case hipSuccess:
                    // TODO return the img_id of the request that was completed.
                    *img_id = this->streams_image_id[i];
                    this->streams_image_id[i] = EMPTY_STREAM;
                    return true;
                case hipErrorNotReady:
                    return false;
                default:
                    CUDA_CHECK(status);
                    return false;
                }
            }
        }

        return false;
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

// TODO implement a lock
// TODO implement a MPMC queue
// TODO implement the persistent kernel
// TODO implement a function for calculating the threadblocks count

class queue_server : public image_processing_server
{
private:
    // TODO define queue server context (memory buffers, etc...)
public:
    queue_server(int threads)
    {
        // TODO initialize host state
        // TODO launch GPU persistent kernel with given number of threads, and calculated number of threadblocks
    }

    ~queue_server() override
    {
        // TODO free resources allocated in constructor
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO push new task into queue if possible
        return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) the producer-consumer queue for any responses.
        return false;

        // TODO return the img_id of the request that was completed.
        //*img_id = ... 
        return true;
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
